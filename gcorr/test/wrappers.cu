#include "hip/hip_runtime.h"
#include <cstddef>
#include <vector>
#include <iostream>
#include <random>

#include <hip/hip_complex.h>

#include "gtest.h"

#include "gpu_array.h"
#include "gxkernel.h"

using std::size_t;

template <typename Wrapped>
double run_kernel(int repeat_count, Wrapped fn) {
    hipEvent_t ev[2];
    hipEventCreate(&ev[0]);
    hipEventCreate(&ev[1]);

    hipEventRecord(ev[0]);
    for (int i = 0; i<repeat_count; ++i) fn();
    hipEventRecord(ev[1]);
    hipEventSynchronize(ev[1]);

    float ms = 0;
    hipEventElapsedTime(&ms, ev[0], ev[1]);
    hipEventDestroy(ev[0]);
    hipEventDestroy(ev[1]);

    return ms/1000.0/(double)repeat_count;
}

inline int nblocks(int n, int width) {
    return n? 1+(n-1)/width: 0;
}

std::vector<float2> run_CrossCorrAccumHoriz(const std::vector<float2>& data, int nant, int nfft, int nchan, int fftwidth) {
    constexpr int npol = 2;

    int block_width = 128;
    dim3 ccblock(nblocks(nchan, block_width), nant-1, nant-1);

    size_t result_sz = nant*(nant-1)/2*npol*npol*nchan;

    gpu_array<float2> gpu_data(data);
    gpu_array<float2> gpu_result(result_sz);

    CrossCorrAccumHoriz<<<ccblock, block_width>>>(gpu_result.data(), gpu_data.data(), nant, nfft, nchan, fftwidth);
    return gpu_result;
}

double time_CrossCorrAccumHoriz(int repeat_count, const float2* gpu_data, int nant, int nfft, int nchan, int fftwidth) {
    constexpr int npol = 2;

    int block_width = 128;
    dim3 ccblock(nblocks(nchan, block_width), nant-1, nant-1);

    size_t result_sz = nant*(nant-1)/2*npol*npol*nchan;
    gpu_array<float2> gpu_result(result_sz);

    return run_kernel(repeat_count,
	[&]() {
	    CrossCorrAccumHoriz<<<ccblock, block_width>>>(gpu_result.data(), gpu_data, nant, nfft, nchan, fftwidth);
	});
}

std::vector<float2> run_CCAH2(const std::vector<float2>& data, int nant, int nfft, int nchan, int fftwidth) {
    constexpr int npol = 2;
    int nantxp = nant*npol;

    int block_width = 128;
    dim3 ccblock(nblocks(nchan, block_width), nantxp-1, nantxp-1);

    size_t result_sz = nant*(nant-1)/2*npol*npol*nchan;

    gpu_array<float2> gpu_data(data);
    gpu_array<float2> gpu_result(result_sz);

    CCAH2<<<ccblock, block_width>>>(gpu_result.data(), gpu_data.data(), nant, nfft, nchan, fftwidth);
    return gpu_result;
}

double time_CCAH2(int repeat_count, const float2* gpu_data, int nant, int nfft, int nchan, int fftwidth) {
    constexpr int npol = 2;
    int nantxp = nant*npol;

    int block_width = 128;
    dim3 ccblock(nblocks(nchan, block_width), nantxp-1, nantxp-1);

    size_t result_sz = nant*(nant-1)/2*npol*npol*nchan;
    gpu_array<float2> gpu_result(result_sz);

    return run_kernel(repeat_count,
	[&]() {
	    CCAH2<<<ccblock, block_width>>>(gpu_result.data(), gpu_data, nant, nfft, nchan, fftwidth);
	});
}

// This is stupidly slow:
#if 0
__global__ void CCAH3(hipComplex *accum, const hipComplex *ants, int nant, int nfft, int nchan, int fftwidth) {
    extern __shared__ float2 h[];

    int t = threadIdx.x+blockIdx.x*blockDim.x;
    if (t>=nchan) return;
    int block_width = blockDim.x;

    // blockIdx.y: index of first vector (2*antennaindex+polindex)

    int s = nfft*fftwidth;

    int i = blockIdx.y;
    int j0 = 2*(i/2+1);

    int ai = i/2;
    int b = 4*(ai*nant-ai*(ai+1)/2) + 2*(i-2*ai);

    const float2* iv = ants+i*s+t;
    const float2* jv = ants+j0*s+t;

    float2 u = iv[0];
    int hoff = threadIdx.x;
    for (int j = j0; j<2*nant; ++j) {
	float2 v = jv[0];
	float2 z;
	z.x = u.x*v.x + u.y*v.y;
	z.y = u.y*v.x - u.x*v.y;
	h[hoff] = z;

	jv += s;
	hoff += block_width;
    }

    for (int k = fftwidth; k<s; k += fftwidth) {
	u = iv[k];
	jv = ants+j0*s+t;

	int hoff = threadIdx.x;
	for (int j = j0; j<2*nant; ++j) {
	    float2 v = jv[k];
	    float2 z;
	    z.x = u.x*v.x + u.y*v.y;
	    z.y = u.y*v.x - u.x*v.y;
	    h[hoff].x += z.x;
	    h[hoff].y += z.y;

	    jv += s;
	    hoff += block_width;
	}
    }

    float oonfft = 1.f/nfft;

    hoff = threadIdx.x;
    for (int j = j0; j<2*nant; ++j) {
	float2 a = h[hoff];
	a.x *= oonfft;
	a.y *= oonfft;

	int dj = j-j0;
	int aj = 2*(dj/2);
	accum[(b+aj*2+(dj-aj))*nchan+t] = a;
	hoff += block_width;
    }
}
#elif 1
constexpr int ccah3_cwidth = 4096;
// Plan:
//  1.  Use shared mem for vector i cache with blockDim.x == ccah3_width.'
//      split vector horizontally in griDim.x blocks, gridDim.x*blockDim.x>=nchan*nfft.
//      For ease of impl, presume nchan|ccah3_width.
//  2.  Each block loads k nchan-blocks for vector i into cache.
//  3.  Horizontal accumulation is performed serially across vectors j; either pre-zero
//      accumulator or run left-side (offset 0) kernel first.

template <bool initial>
__global__ void CCAH3(hipComplex *accum, const hipComplex *ants, int nant, int nfft, int nchan, int fftwidth) {
    extern __shared__ float2 h[ccah3_cwidth];

    assert(ccah_cwidth%nchan==0);
    assert(
    int t = threadIdx.x;
    int x = 0;
    if (initial) {
	assert(blockIdx.x==0);
    }
    else {
	x = fftwidth*(blockIdx.x+1); // second invocation: run nchan*nfft/cwidth-1 blocks.
    }

    if (t>=nchan) return;
    int block_width = blockDim.x;

    // blockIdx.y: index of first vector (2*antennaindex+polindex)

    int s = nfft*fftwidth;

    int i = blockIdx.y;
    int j0 = 2*(i/2+1);

    int ai = i/2;
    int b = 4*(ai*nant-ai*(ai+1)/2) + 2*(i-2*ai);

    const float2* iv = ants+i*s+t;
    const float2* jv = ants+j0*s+t;

    float2 u = iv[0];
    int hoff = threadIdx.x;
    for (int j = j0; j<2*nant; ++j) {
	float2 v = jv[0];
	float2 z;
	z.x = u.x*v.x + u.y*v.y;
	z.y = u.y*v.x - u.x*v.y;
	h[hoff] = z;

	jv += s;
	hoff += block_width;
    }

    for (int k = fftwidth; k<s; k += fftwidth) {
	u = iv[k];
	jv = ants+j0*s+t;

	int hoff = threadIdx.x;
	for (int j = j0; j<2*nant; ++j) {
	    float2 v = jv[k];
	    float2 z;
	    z.x = u.x*v.x + u.y*v.y;
	    z.y = u.y*v.x - u.x*v.y;
	    h[hoff].x += z.x;
	    h[hoff].y += z.y;

	    jv += s;
	    hoff += block_width;
	}
    }

    float oonfft = 1.f/nfft;

    hoff = threadIdx.x;
    for (int j = j0; j<2*nant; ++j) {
	float2 a = h[hoff];
	a.x *= oonfft;
	a.y *= oonfft;

	int dj = j-j0;
	int aj = 2*(dj/2);
	accum[(b+aj*2+(dj-aj))*nchan+t] = a;
	hoff += block_width;
    }
template <int pj>
__global__ void CCAH3(hipComplex *accum, const hipComplex *ants, int nant, int nfft, int nchan, int fftwidth) {
}
#endif

std::vector<float2> run_CCAH3(const std::vector<float2>& data, int nant, int nfft, int nchan, int fftwidth) {
    constexpr int npol = 2;
    int nantxp = nant*npol;

    int block_width = 128;
    dim3 ccblock(nblocks(nchan, block_width), nantxp-1);

    size_t result_sz = nant*(nant-1)/2*npol*npol*nchan;

    gpu_array<float2> gpu_data(data);
    gpu_array<float2> gpu_result(result_sz);

    int shared_alloc =sizeof(float2)*(nantxp-2)*block_width;
    CCAH3<<<ccblock, block_width, shared_alloc>>>(gpu_result.data(), gpu_data.data(), nant, nfft, nchan, fftwidth);
    return gpu_result;
}

double time_CCAH3(int repeat_count, const float2* gpu_data, int nant, int nfft, int nchan, int fftwidth) {
    constexpr int npol = 2;
    int nantxp = nant*npol;

    int block_width = 128;
    dim3 ccblock(nblocks(nchan, block_width), nantxp-1);

    size_t result_sz = nant*(nant-1)/2*npol*npol*nchan;
    gpu_array<float2> gpu_result(result_sz);

    int shared_alloc =sizeof(float2)*(nantxp-2)*block_width;
    return run_kernel(repeat_count,
	[&]() {
	    CCAH3<<<ccblock, block_width, shared_alloc>>>(gpu_result.data(), gpu_data, nant, nfft, nchan, fftwidth);
	});
}

std::vector<float2> run_CrossCorr(const std::vector<float2>& data, int nant, int nfft, int nchan, int fftwidth) {
    int targetThreads = 50e4;
    int parallelAccum = (int)ceil(targetThreads/nchan+1);
    while (parallelAccum && nfft % parallelAccum) parallelAccum--;

    int block_width = 512;
    int blockx = nblocks(nchan, block_width);

    dim3 corrBlocks(blockx, parallelAccum);
    dim3 accumBlocks(blockx, 4, nant*(nant-1)/2);

    size_t result_sz = nant*(nant-1)*2*nchan;

    gpu_array<float2> gpu_data(data);
    gpu_array<float2> gpu_baselinedata(result_sz*parallelAccum);

    int nchunk = nfft/parallelAccum;
    CrossCorr<<<corrBlocks, block_width>>>(gpu_data.data(), gpu_baselinedata.data(), nant, nchunk);
    finaliseAccum<<<accumBlocks, block_width>>>(gpu_baselinedata.data(), parallelAccum, nchunk);

    std::vector<float2> baselinedata(gpu_baselinedata);
    std::vector<float2> result(result_sz);

    int nvec = nant*(nant-1)*2;
    int rstride = nchan, bstride = nchan*parallelAccum;
    for (int i = 0; i<nvec; ++i) {
        std::copy(baselinedata.data()+i*bstride, baselinedata.data()+i*bstride+nchan, result.data()+i*rstride);
    }

    return result;
}

double time_CrossCorr(int repeat_count, const float2* gpu_data, int nant, int nfft, int nchan, int fftwidth) {
    int targetThreads = 50e4;
    int parallelAccum = (int)ceil(targetThreads/nchan+1);
    while (parallelAccum && nfft % parallelAccum) parallelAccum--;

    int block_width = 512;
    int blockx = nblocks(nchan, block_width);

    dim3 corrBlocks(blockx, parallelAccum);
    dim3 accumBlocks(blockx, 4, nant*(nant-1)/2);

    size_t result_sz = nant*(nant-1)*2*nchan;
    gpu_array<float2> gpu_baselinedata(result_sz*parallelAccum);

    return run_kernel(repeat_count,
	[&]() {
            CrossCorr<<<corrBlocks, block_width>>>((float2*)gpu_data, gpu_baselinedata.data(), nant, nfft/parallelAccum);
            finaliseAccum<<<accumBlocks, block_width>>>(gpu_baselinedata.data(), parallelAccum, nfft/parallelAccum);
        });
}

// No unit test for FringeRotate routines yet.

double time_FringeRotate(int repeat_count, const float2* gpu_data, const float* gpu_rotvec, int nant, int nfft, int fftwidth) {
    int block_width = 512;
    dim3 fringeBlocks(nblocks(fftwidth, block_width), nfft);

    return run_kernel(repeat_count,
	[&]() {
            FringeRotate<<<fringeBlocks, block_width>>>((float2*)gpu_data, (float*)gpu_rotvec);
        });
}

double time_FringeRotate2(int repeat_count, const float2* gpu_data, const float* gpu_rotvec, int nant, int nfft, int fftwidth) {
    int block_width = 512;
    dim3 fringeBlocks(nblocks(fftwidth, block_width), nfft);

    return run_kernel(repeat_count,
	[&]() {
            FringeRotate2<<<fringeBlocks, block_width>>>((float2*)gpu_data, (float*)gpu_rotvec);
        });

}
