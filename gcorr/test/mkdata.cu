#include "hip/hip_runtime.h"
#include <cstddef>
#include <vector>
#include <algorithm>
#include <random>

#include <hip/hip_complex.h>

#include "mkdata.h"

// Use complex small magnitude ints to avoid rouding artefacts in testing.
std::vector<float2> random_cint_data(std::size_t n, int vmin, int vmax, int seed) {
    std::minstd_rand R(seed);
    std::uniform_int_distribution<int> U(vmin, vmax);

    std::vector<float2> data(n);
    std::generate(data.begin(), data.end(), [&]() {
	float2 f;
	f.x = U(R);
	f.y = U(R);
       	return f;
    });
    return data;
}

std::vector<float2> random_cfloat_data(std::size_t n, float vmin, float vmax, int seed) {
    std::minstd_rand R(seed);
    std::uniform_real_distribution<float> U(vmin, vmax);

    std::vector<float2> data(n);
    std::generate(data.begin(), data.end(), [&]() {
	float2 f;
	f.x = U(R);
	f.y = U(R);
       	return f;
    });
    return data;
}

std::vector<float> random_float_data(std::size_t n, float vmin, float vmax, int seed) {
    std::minstd_rand R(seed);
    std::uniform_real_distribution<float> U(vmin, vmax);

    std::vector<float> data(n);
    std::generate(data.begin(), data.end(), [&]() { return U(R); });
    return data;
}

std::vector<int32_t> random_int_data(std::size_t n, int vmin, int vmax, int seed) {
    std::minstd_rand R(seed);
    std::uniform_int_distribution<int32_t> U(vmin, vmax);

    std::vector<int32_t> data(n);
    std::generate(data.begin(), data.end(), [&]() { return U(R); });
    return data;
}

std::vector<int8_t> random_int8_data(std::size_t n, int8_t vmin, int8_t vmax, int seed) {
    std::minstd_rand R(seed);
    std::uniform_int_distribution<int8_t> U(vmin, vmax);

    std::vector<int8_t> data(n);
    std::generate(data.begin(), data.end(), [&]() { return U(R); });
    return data;
}
